//device code
#include <stdlib.h>  
#include <hip/hip_runtime.h> 
#include <stdio.h>  

//设备端代码
__global__ void cvlUnit(const char *imgR,const char *imgG,const char *imgB,const char *core,char *outR,char *outG,char *outB,int lenX,int lenY,int lenCore)  
{  
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int index = x*lenX+y;
	//judgement of index
	if(x<0|x>=lenX|y<0|y>=lenY){
		return;
	}
	//judgement of boundary,return value directly
	if(x-lenCore/2<0|x+lenCore/2>=lenX|y-lenCore/2<0|y+lenCore/2>=lenY){
		outR[index]=imgR[index];
		outG[index]=imgG[index];
		outB[index]=imgB[index];
		return;
	}
	int i,j;
	int tmpX,tmpY;
	int sumR=0,sumG=0,sumB=0;
	for(i=0;i<lenCore;i++){
		for(j=0;j<lenCore;j++){
			tmpX = x-lenCore/2+i;
			tmpY = y-lenCore/2+j;
			sumR += imgR[tmpX*lenX+tmpY]*core[i*lenCore+j];
			sumG += imgG[tmpX*lenX+tmpY]*core[i*lenCore+j];
			sumB += imgB[tmpX*lenX+tmpY]*core[i*lenCore+j];
		}
	}
	outR[index]=(char)(sumR*1.0/(lenCore*lenCore));
	outG[index]=(char)(sumG*1.0/(lenCore*lenCore));
	outB[index]=(char)(sumB*1.0/(lenCore*lenCore));
	return;
}  
