#include "hip/hip_runtime.h"
#include <stdlib.h>  
#include <hip/hip_runtime.h> 
#include <stdio.h>  
#include "heyD.cu"
void display(int*,int);
__global__ void matrixAdd(const int *, const int *, int *, int);  
//主机端代码
int func() // 注意这里定义形式  
{
	int num = 4;
	int size = num*num*sizeof(int);
	int *matrixA=(int*)malloc(size);
	int *matrixB=(int*)malloc(size);
	int i=0;
	for (i=0;i<num*num;i++){
		matrixA[i]=i+1;
		matrixB[i]=2*(i+1);
	}
	int *matrixC=(int*)malloc(size);

	int *matrixAd=NULL,*matrixBd=NULL,*matrixCd=NULL;

	hipMalloc((void**)&matrixAd,size);
	hipMalloc((void**)&matrixBd,size);
	hipMalloc((void**)&matrixCd,size);
	hipMemcpy(matrixAd,matrixA,size,hipMemcpyHostToDevice);
	hipMemcpy(matrixBd,matrixB,size,hipMemcpyHostToDevice);

	int blockX = 2;int blockY = 2;
	dim3 dimGrid(num/blockX,num/blockY);
	dim3 dimBlock(blockX,blockY);
	matrixAdd<<<dimGrid,dimBlock>>>(matrixAd,matrixBd,matrixCd,num);

	hipMemcpy(matrixC,matrixCd,size,hipMemcpyDeviceToHost);
	printf("matrixA:\n");display(matrixA,num);
	printf("matrixB:\n");display(matrixB,num);
	printf("matrixC:\n");display(matrixC,num);
	return 0;
}
void display(int* a,int num){
	int i=0,j=0;
	int index=0;
	for(i=0;i<num;i++){
		for(j=0;j<num;j++){
			index=i*num+j;
			printf("\t%2d",a[index]);
		}
		printf("\n");
	}
}







