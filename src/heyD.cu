#include <stdlib.h>  
#include <hip/hip_runtime.h> 
#include <stdio.h>  

//设备端代码
__global__ void matrixAdd(const int *A, const int *B, int *C, int num)  
{  
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	C[x * num + y] = A[x * num + y] + B[x * num + y];
	return;
}  
