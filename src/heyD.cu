#include <stdlib.h>  
#include <hip/hip_runtime.h> 
#include <stdio.h>  

//设备端代码
__global__ void matrixAdd(const int *A, const int *B, int *C, int num)  
{  
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int i,j;
	for(i=0;i<num;i++){
		for(j=0;j<num;j++){
			C[x*num+y]+=A[i*num+j]*B[x*num+y];
		}
	}
	//	C[x * num + y] = A[x * num + y] + B[x * num + y];
	return;
}  
