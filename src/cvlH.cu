#include "hip/hip_runtime.h"
//host code
#include <stdlib.h>  
#include <hip/hip_runtime.h> 
#include <stdio.h>  
#include <opencv2/opencv.hpp>
#include "cvlD.cu"
using namespace cv;
__global__ void cvlUnit(const char *imgR,const char *imgG,const char *imgB,const char *core,char *outR,char *outG,char *outB,int lenX,int lenY,int lenCore);  
void mat2pointerCore(Mat core,char* coreH);
void mat2pointerImg(Mat img,int z,char* imgH);
Mat pointerToMat(char* r,char* g,char* b,Mat img);
//机端代码
Mat cudaCvl(Mat img,Mat core){
	int lenX = img.cols;
	int lenY = img.rows; 
	int lenCore = core.cols;
	int lenBlock = 16;
	int lenGridX = lenX/lenBlock;
	int lenGridY = lenY/lenBlock;
	if(lenBlock*lenGridX!=lenX)lenGridX++;
	if(lenBlock*lenGridY!=lenX)lenGridY++;

	int sizeOfImage = lenX*lenY*sizeof(char);
	int sizeOfCore = lenCore*lenCore*sizeof(char);

	//host data define
	char* imgRH = (char*)malloc(sizeOfImage);
	char* imgGH = (char*)malloc(sizeOfImage);
	char* imgBH = (char*)malloc(sizeOfImage);
	char* coreH = (char*)malloc(sizeOfCore);
	mat2pointerImg(img,0,imgBH);
	mat2pointerImg(img,1,imgGH);
	mat2pointerImg(img,2,imgRH);
	mat2pointerCore(core,coreH);

	//device data define
	char *imgRD=NULL,*imgGD=NULL,*imgBD=NULL,*coreD=NULL;
	hipMalloc((void**)&imgRD,sizeOfImage);
	hipMalloc((void**)&imgGD,sizeOfImage);
	hipMalloc((void**)&imgBD,sizeOfImage);
	hipMalloc((void**)&coreD,sizeOfCore);
	hipMemcpy(imgRD,imgRH,sizeOfImage,hipMemcpyHostToDevice);
	hipMemcpy(imgGD,imgGH,sizeOfImage,hipMemcpyHostToDevice);
	hipMemcpy(imgBD,imgBH,sizeOfImage,hipMemcpyHostToDevice);
	hipMemcpy(coreD,coreH,sizeOfCore,hipMemcpyHostToDevice);
	char *outRD=NULL,*outGD=NULL,*outBD=NULL;
	hipMalloc((void**)&outRD,sizeOfImage);
	hipMalloc((void**)&outRD,sizeOfImage);
	hipMalloc((void**)&outRD,sizeOfImage);

	//start work
	dim3 dimGrid(lenGridX,lenGridY);
	dim3 dimBlock(lenBlock,lenBlock);
	cvlUnit<<<dimGrid,dimBlock>>>(imgRD,imgGD,imgBD,coreD,outRD,outGD,outBD,lenX,lenY,lenCore);


	hipMemcpy(imgRH,outRD,sizeOfImage,hipMemcpyDeviceToHost);
	hipMemcpy(imgGH,outGD,sizeOfImage,hipMemcpyDeviceToHost);
	hipMemcpy(imgBH,outBD,sizeOfImage,hipMemcpyDeviceToHost);
	Mat result = pointerToMat(imgRH,imgGH,imgBH,img);
	return result;
} 
Mat pointerToMat(char* r,char* g,char* b,Mat img){
	Mat result = img.clone();
	int lenX = img.cols;
	int lenY = img.rows;
	int i,j;
	for(i=0;i<lenX;i++){
		for(j=0;j<lenY;j++){
			result.at<Vec3b>(i,j)[0]=b[i*lenX+j];
			result.at<Vec3b>(i,j)[1]=g[i*lenX+j];
			result.at<Vec3b>(i,j)[2]=r[i*lenX+j];
		}
	}
	return result;
}
void mat2pointerImg(Mat img,int z,char* imgH){
	int lenX = img.cols;
	int lenY = img.rows;
	int i,j;
	for(i=0;i<lenX;i++){
		for(j=0;j<lenY;j++){
			imgH[i*lenX+j]=img.at<Vec3b>(i,j)[z];
		}
	}
	return;
}
void mat2pointerCore(Mat core,char* coreH){
	int lenCore = core.rows;
	int i,j;
	for(i=0;i<lenCore;i++){
		for(j=0;j>lenCore;j++){
			coreH[i*lenCore+j]=core.at<char>(i,j);
		}		
	}
	return;
}







