#include "hip/hip_runtime.h"
//host code
#include <stdlib.h>  
#include <hip/hip_runtime.h> 
#include <stdio.h>  
#include <opencv2/opencv.hpp>
#include "cvlD.cu"
using namespace cv;
__global__ void cvlUnit(const char *imgR,const char *imgG,const char *imgB,const char *core,char *outR,char *outG,char *outB,int lenX,int lenY,int lenCore);  
void initCvlUnit(char* imgRH,char* imgGH,char* imgBH,char* coreH,int lenX,int lenY,int lenCore,char* imgRD,char*imgGD,char* imgBD,char* coreD,char* outRD,char* outGD,char* outBD);
void mat2pointerCore(Mat core,char* coreH);
void mat2pointerImg(Mat img,int z,char* imgH);
void display(char*,int);
Mat pointerToMat(char* r,char* g,char* b,Mat img);
//机端代码
Mat cudaCvl(Mat img,Mat core){
	int lenX = img.cols;
	int lenY = img.rows; 
	int lenCore = core.cols;
	
//	printf("%d,%d,%d\n",lenX,lenY,lenGridX);
	int sizeOfImg = lenX*lenY*sizeof(char);
	int sizeOfCore = lenCore*lenCore*sizeof(char);


	//cuda malloc have to be done first,but i do not know why
	char *imgRD=NULL,*imgGD=NULL,*imgBD=NULL,*coreD=NULL,*outRD=NULL,*outGD=NULL,*outBD=NULL;
	hipMalloc((void**)&imgRD,sizeOfImg);
	hipMalloc((void**)&imgGD,sizeOfImg);
	hipMalloc((void**)&imgBD,sizeOfImg);
	hipMalloc((void**)&coreD,sizeOfCore);
	hipMalloc((void**)&outRD,sizeOfImg);
	hipMalloc((void**)&outGD,sizeOfImg);
	hipMalloc((void**)&outBD,sizeOfImg);

	//host data define
	char* imgRH = (char*)malloc(sizeOfImg);
	char* imgGH = (char*)malloc(sizeOfImg);
	char* imgBH = (char*)malloc(sizeOfImg);
	char* coreH = (char*)malloc(sizeOfCore);
	mat2pointerImg(img,0,imgBH);
	mat2pointerImg(img,1,imgGH);
	mat2pointerImg(img,2,imgRH);
	mat2pointerCore(core,coreH);


	initCvlUnit(imgRH,imgGH,imgBH,coreH,lenX,lenY,lenCore,imgRD,imgGD,imgBD,coreD,outRD,outGD,outBD);

	Mat result = pointerToMat(imgRH,imgGH,imgBH,img);
	return result;
} 
void initCvlUnit(char* imgRH,char* imgGH,char* imgBH,char* coreH,int lenX,int lenY,int lenCore,char* imgRD,char*imgGD,char* imgBD,char* coreD,char* outRD,char* outGD,char* outBD){
	int sizeOfImg = lenX*lenY*sizeof(char);
	int sizeOfCore = lenCore*lenCore*sizeof(char);
	char *outRH=(char*)malloc(sizeOfImg);
	char *outGH=(char*)malloc(sizeOfImg);
	char *outBH=(char*)malloc(sizeOfImg);
	hipMemcpy(imgRD,imgRH,sizeOfImg,hipMemcpyHostToDevice);
	hipMemcpy(imgGD,imgGH,sizeOfImg,hipMemcpyHostToDevice);
	hipMemcpy(imgBD,imgBH,sizeOfImg,hipMemcpyHostToDevice);
	hipMemcpy(coreD,coreH,sizeOfCore,hipMemcpyHostToDevice);

	int lenBlock =16;
	int lenGridX = lenX/lenBlock;
	int lenGridY = lenY/lenBlock;
	if(lenBlock*lenGridX!=lenX)lenGridX++;
	if(lenBlock*lenGridY!=lenX)lenGridY++;
	dim3 dimGrid(lenGridX,lenGridY);
	dim3 dimBlock(lenBlock,lenBlock);
//	printf("  r:\n");display(imgRH,lenX);
//	printf("  g:\n");display(imgGH,lenX);
//	printf("  b:\n");display(imgBH,lenX);
//	printf("  core:\n");display(coreH,lenCore);
//	printf("  %d,%d,%d,%d,%d,%d,%d\n",lenGridX,lenGridY,lenBlock,lenBlock,lenX,lenY,lenCore);
	cvlUnit<<<dimGrid,dimBlock>>>(imgRD,imgGD,imgBD,coreD,outRD,outGD,outBD,lenX,lenY,lenCore);

	hipMemcpy(outRH,outRD,sizeOfImg,hipMemcpyDeviceToHost);
	hipMemcpy(outGH,outGD,sizeOfImg,hipMemcpyDeviceToHost);
	hipMemcpy(outBH,outBD,sizeOfImg,hipMemcpyDeviceToHost);
//	printf("imgRH:\n");display(imgRH,lenX);
//	printf("imgGH:\n");display(imgGH,lenX);
//	printf("imgBH:\n");display(imgBH,lenX);
//	printf("coreH:\n");display(coreH,lenCore);
//	printf("outRH:\n");display(outRH,lenX);
//	printf("outGH:\n");display(outGH,lenX);
//	printf("outBH:\n");display(outBH,lenX);

	hipMemcpy(imgRH,outRH,sizeOfImg,hipMemcpyHostToHost);
	hipMemcpy(imgGH,outGH,sizeOfImg,hipMemcpyHostToHost);
	hipMemcpy(imgBH,outBH,sizeOfImg,hipMemcpyHostToHost);
	return;
}
Mat pointerToMat(char* r,char* g,char* b,Mat img){
	Mat result = img.clone();
	int lenX = img.cols;
	int lenY = img.rows;
	int i,j;
	for(i=0;i<lenX;i++){
		for(j=0;j<lenY;j++){
			result.at<Vec3b>(i,j)[0]=b[i*lenX+j];
			result.at<Vec3b>(i,j)[1]=g[i*lenX+j];
			result.at<Vec3b>(i,j)[2]=r[i*lenX+j];
		}
	}
	return result;
}
void mat2pointerImg(Mat img,int z,char* imgH){
	int lenX = img.cols;
	int lenY = img.rows;
	int i,j;
	for(i=0;i<lenX;i++){
		for(j=0;j<lenY;j++){
			imgH[i*lenX+j]=img.at<Vec3b>(i,j)[z];
		}
	}
	return;
}
void mat2pointerCore(Mat core,char* coreH){
	int lenCore = core.rows;
	int i,j;
	for(i=0;i<lenCore;i++){
		for(j=0;j<lenCore;j++){
			coreH[i*lenCore+j]=core.at<char>(i,j);
		}		
	}
	return;
}

void display(char* a,int num){
	int i=0,j=0;
	int index=0;
	for(i=0;i<num;i++){
		for(j=0;j<num;j++){
			index=i*num+j;
			printf("\t%2d",a[index]);
		}
		printf("\n");
	}
}






